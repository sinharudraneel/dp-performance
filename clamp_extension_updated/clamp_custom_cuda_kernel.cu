#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include <vector>

template <typename scalar_t>
__global__ void clamp_custom_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int64_t* __restrict__ sizes,
    const int64_t* __restrict__ strides,
    int64_t numel,
    int ndim,
    scalar_t min_val,
    scalar_t max_val
) {
    //if (threadIdx.x == 0 && blockIdx.x == 0) {
    //	printf("ndim: %lld\n", ndim);
    //	printf("")
    //}
    const int64_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < numel) {
        int64_t input_index = 0;
        int64_t remaining = index;
        for (int i = 0; i < ndim; ++i) {
            int64_t coord = remaining / strides[i];
            remaining %= strides[i];
            input_index += coord * strides[i];
        }
        scalar_t val = input[input_index];
        output[input_index] = max(min(val, max_val), min_val);
    }
}

template <typename scalar_t>
void launch_clamp_custom_kernel(
    const scalar_t* input,
    scalar_t* output,
    const int64_t* sizes,
    const int64_t* strides,
    int64_t numel,
    int ndim,
    scalar_t min_val,
    scalar_t max_val
) {
    const int threads = 1024;
    const int blocks = (numel + threads - 1) / threads;

    auto d_sizes_tensor = torch::from_blob(const_cast<int64_t*>(sizes), {ndim},
		    	  torch::TensorOptions().dtype(torch::kInt64))
	    		  .to(torch::kCUDA);
    auto d_strides_tensor = torch::from_blob(const_cast<int64_t*>(strides), {ndim},
		    	    torch::TensorOptions().dtype(torch::kInt64))
	    		    .to(torch::kCUDA);
    clamp_custom_kernel<<<blocks, threads>>>(
        input, output, d_sizes_tensor.data_ptr<int64_t>(), d_strides_tensor.data_ptr<int64_t*>(), numel, ndim, min_val, max_val);

    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
    	throw std::runtime_error(hipGetErrorString(err));
    }
}

torch::Tensor clamp_custom_cuda(
    torch::Tensor input,
    double min_val,
    double max_val
) {
    auto output = torch::empty_like(input);
    auto sizes = input.sizes().vec();
    auto strides = input.strides().vec();

    if (input.numel() == 0) {
    	return input;
    }
    if (min_val > max_val) {
    	throw std::runtime_error("min/max values are invalid (min greater than max)");
    }

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "clamp_custom_cuda", ([&] {
        launch_clamp_custom_kernel<scalar_t>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            sizes.data(),
            strides.data(),
            input.numel(),
            input.dim(),
            static_cast<scalar_t>(min_val),
            static_cast<scalar_t>(max_val)
        );
    }));
    return output;
}
