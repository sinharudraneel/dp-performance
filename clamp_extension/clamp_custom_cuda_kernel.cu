#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include <vector>

template <typename scalar_t>
__global__ void clamp_custom_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int64_t* __restrict__ sizes,
    const int64_t* __restrict__ strides,
    int64_t numel,
    int ndim,
    scalar_t min_val,
    scalar_t max_val
) {
    //if (threadIdx.x == 0 && blockIdx.x == 0) {
    //	printf("ndim: %lld\n", ndim);
    //	printf("")
    //}
    const int64_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < numel) {
        int64_t input_index = 0;
        int64_t remaining = index;
        for (int i = 0; i < ndim; ++i) {
            int64_t coord = remaining / strides[i];
            remaining %= strides[i];
            input_index += coord * strides[i];
        }
        scalar_t val = input[input_index];
        output[input_index] = max(min(val, max_val), min_val);
    }
}

template <typename scalar_t>
void launch_clamp_custom_kernel(
    const scalar_t* input,
    scalar_t* output,
    const int64_t* sizes,
    const int64_t* strides,
    int64_t numel,
    int ndim,
    scalar_t min_val,
    scalar_t max_val
) {
    const int threads = 1024;
    const int blocks = (numel + threads - 1) / threads;

    int64_t *d_sizes, *d_strides;
    hipMalloc(&d_sizes, ndim * sizeof(int64_t));
    hipMalloc(&d_strides, ndim * sizeof(int64_t));
    hipMemcpy(d_sizes, sizes, ndim * sizeof(int64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_strides, strides, ndim * sizeof(int64_t), hipMemcpyHostToDevice);

    //std::vector<int64_t> h_sizes(ndim), h_strides(ndim);
    //hipMemcpy(h_sizes.data(), d_sizes, ndim * sizeof(int64_t), hipMemcpyDeviceToHost);
    //hipMemcpy(h_strides.data(), d_strides, ndim * sizeof(int64_t), hipMemcpyDeviceToHost);
    
    //std::cout << "Sizes and strides on GPU:" << std::endl;
    //for (int i = 0; i < ndim; ++i) {
    //    std::cout << "sizes[" << i << "]: " << h_sizes[i] 
    //              << ", strides[" << i << "]: " << h_strides[i] << std::endl;
    //}

    clamp_custom_kernel<<<blocks, threads>>>(
        input, output, d_sizes, d_strides, numel, ndim, min_val, max_val);

    hipFree(d_sizes);
    hipFree(d_strides);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
    	throw std::runtime_error(hipGetErrorString(err));
    }
}

torch::Tensor clamp_custom_cuda(
    torch::Tensor input,
    double min_val,
    double max_val
) {
    auto output = torch::empty_like(input);
    auto sizes = input.sizes().vec();
    auto strides = input.strides().vec();

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "clamp_custom_cuda", ([&] {
        launch_clamp_custom_kernel<scalar_t>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            sizes.data(),
            strides.data(),
            input.numel(),
            input.dim(),
            static_cast<scalar_t>(min_val),
            static_cast<scalar_t>(max_val)
        );
    }));
    return output;
}
